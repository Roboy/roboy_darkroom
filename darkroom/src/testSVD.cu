#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>

int main(void)
{
    // H has storage for 4 integers
    thrust::host_vector<int> H(4);

    // initialize individual elements
    H[0] = 14;
    H[1] = 20;
    H[2] = 38;
    H[3] = 46;

    // H.size() returns the size of vector H
    std::cout << "H has size " << H.size() << std::endl;

    // print contents of H
    for(int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << std::endl;

    // resize H
    H.resize(2);

    std::cout << "H now has size " << H.size() << std::endl;

    // Copy host_vector H to device_vector D
    thrust::device_vector<int> D = H;

    // elements of D can be modified
    D[0] = 99;
    D[1] = 88;

    // print contents of D
    for(int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // H and D are automatically deleted when the function returns
    return 0;
}

////Example 2. Application Using C and CUBLAS: 0-based indexing
////-----------------------------------------------------------
//#include <stdio.h>
//#include <stdlib.h>
//#include <math.h>
//#include <hip/hip_runtime.h>
//#include "hipblas.h"
//#define M 6
//#define N 5
//#define IDX2C(i,j,ld) (((j)*(ld))+(i))
//
//static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
//    hipblasSscal (handle, n-p, &alpha, &m[IDX2C(p,q,ldm)], ldm);
//    hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
//}
//
//int main (void){
//    hipError_t cudaStat;
//    hipblasStatus_t stat;
//    hipblasHandle_t handle;
//    int i, j;
//    float* devPtrA;
//    float* a = 0;
//    a = (float *)malloc (M * N * sizeof (*a));
//    if (!a) {
//        printf ("host memory allocation failed");
//        return EXIT_FAILURE;
//    }
//    for (j = 0; j < N; j++) {
//        for (i = 0; i < M; i++) {
//            a[IDX2C(i,j,M)] = (float)(i * M + j + 1);
//        }
//    }
//    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
//    if (cudaStat != hipSuccess) {
//        printf ("device memory allocation failed");
//        return EXIT_FAILURE;
//    }
//    stat = hipblasCreate(&handle);
//    if (stat != HIPBLAS_STATUS_SUCCESS) {
//        printf ("CUBLAS initialization failed\n");
//        return EXIT_FAILURE;
//    }
//    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
//    if (stat != HIPBLAS_STATUS_SUCCESS) {
//        printf ("data download failed");
//        hipFree (devPtrA);
//        hipblasDestroy(handle);
//        return EXIT_FAILURE;
//    }
//    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);
//    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
//    if (stat != HIPBLAS_STATUS_SUCCESS) {
//        printf ("data upload failed");
//        hipFree (devPtrA);
//        hipblasDestroy(handle);
//        return EXIT_FAILURE;
//    }
//    hipFree (devPtrA);
//    hipblasDestroy(handle);
//    for (j = 0; j < N; j++) {
//        for (i = 0; i < M; i++) {
//            printf ("%7.0f", a[IDX2C(i,j,M)]);
//        }
//        printf ("\n");
//    }
//    free(a);
//    return EXIT_SUCCESS;
//}







///*
// * How to compile (assume cuda is installed at /usr/local/cuda/)
// *   nvcc -c -I/usr/local/cuda/include svd_example.cpp
// *   g++ -fopenmp -o a.out svd_example.o -L/usr/local/cuda/lib64 -lcudart -lcublas -lcusolver
// *
// */
//
//#include <stdio.h>
//#include <stdlib.h>
//#include <string.h>
//#include <assert.h>
//#include <hip/hip_runtime.h>
//#include <hipblas.h>
//#include <hipsolver.h>
//
//void printMatrix(int m, int n, const double*A, int lda, const char* name)
//{
//    for(int row = 0 ; row < m ; row++){
//        for(int col = 0 ; col < n ; col++){
//            double Areg = A[row + col*lda];
//            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
//        }
//    }
//}
//
//int main(int argc, char*argv[])
//{
//    hipsolverHandle_t cusolverH = NULL;
//    hipblasHandle_t cublasH = NULL;
//    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
//    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
//    hipError_t cudaStat1 = hipSuccess;
//    hipError_t cudaStat2 = hipSuccess;
//    hipError_t cudaStat3 = hipSuccess;
//    hipError_t cudaStat4 = hipSuccess;
//    hipError_t cudaStat5 = hipSuccess;
//    hipError_t cudaStat6 = hipSuccess;
//    const int m = 3;
//    const int n = 2;
//    const int lda = m;
///*       | 1 2  |
// *   A = | 4 5  |
// *       | 2 1  |
// */
//    double A[lda*n] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
//    double U[lda*m]; // m-by-m unitary matrix
//    double VT[lda*n];  // n-by-n unitary matrix
//    double S[n]; // singular value
//    double S_exact[n] = {7.065283497082729, 1.040081297712078};
//
//    double *d_A = NULL;
//    double *d_S = NULL;
//    double *d_U = NULL;
//    double *d_VT = NULL;
//    int *devInfo = NULL;
//    double *d_work = NULL;
//    double *d_rwork = NULL;
//    double *d_W = NULL;  // W = S*VT
//
//    int lwork = 0;
//    int info_gpu = 0;
//    const double h_one = 1;
//    const double h_minus_one = -1;
// printf("A = (matlab base-1)\n");
//    printMatrix(m, n, A, lda, "A");
//    printf("=====\n");
//
//// step 1: create cusolverDn/cublas handle
//    cusolver_status = hipsolverDnCreate(&cusolverH);
//    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
//
//    cublas_status = hipblasCreate(&cublasH);
//    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
//
//// step 2: copy A and B to device
//    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(double)*lda*n);
//    cudaStat2 = hipMalloc ((void**)&d_S  , sizeof(double)*n);
//    cudaStat3 = hipMalloc ((void**)&d_U  , sizeof(double)*lda*m);
//    cudaStat4 = hipMalloc ((void**)&d_VT , sizeof(double)*lda*n);
//    cudaStat5 = hipMalloc ((void**)&devInfo, sizeof(int));
//    cudaStat6 = hipMalloc ((void**)&d_W  , sizeof(double)*lda*n);
//    assert(hipSuccess == cudaStat1);
//    assert(hipSuccess == cudaStat2);
//    assert(hipSuccess == cudaStat3);
//    assert(hipSuccess == cudaStat4);
//    assert(hipSuccess == cudaStat5);
//    assert(hipSuccess == cudaStat6);
//
//    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*n, hipMemcpyHostToDevice);
//    assert(hipSuccess == cudaStat1);
//
//// step 3: query working space of SVD
//    cusolver_status = hipsolverDnDgesvd_bufferSize(
//        cusolverH,
//        m,
//        n,
//        &lwork );
//    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
//
//    cudaStat1 = hipMalloc((void**)&d_work , sizeof(double)*lwork);
//    assert(hipSuccess == cudaStat1);
//
//// step 4: compute SVD
//    signed char jobu = 'A'; // all m columns of U
//    signed char jobvt = 'A'; // all n columns of VT
//    cusolver_status = hipsolverDnDgesvd (
//        cusolverH,
//        jobu,
//        jobvt,
//        m,
//        n,
//        d_A,
//        lda,
//        d_S,
//        d_U,
//        lda,  // ldu
//        d_VT,
//        lda, // ldvt,
//        d_work,
//        lwork,
//        d_rwork,
//        devInfo);
//    cudaStat1 = hipDeviceSynchronize();
//    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
//    assert(hipSuccess == cudaStat1);
//
//      cudaStat1 = hipMemcpy(U , d_U , sizeof(double)*lda*m, hipMemcpyDeviceToHost);
//    cudaStat2 = hipMemcpy(VT, d_VT, sizeof(double)*lda*n, hipMemcpyDeviceToHost);
//    cudaStat3 = hipMemcpy(S , d_S , sizeof(double)*n    , hipMemcpyDeviceToHost);
//    cudaStat4 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
//    assert(hipSuccess == cudaStat1);
//    assert(hipSuccess == cudaStat2);
//    assert(hipSuccess == cudaStat3);
//    assert(hipSuccess == cudaStat4);
//
//    printf("after gesvd: info_gpu = %d\n", info_gpu);
//    assert(0 == info_gpu);
//    printf("=====\n");
//
//    printf("S = (matlab base-1)\n");
//    printMatrix(n, 1, S, lda, "S");
//    printf("=====\n");
//
//    printf("U = (matlab base-1)\n");
//    printMatrix(m, m, U, lda, "U");
//    printf("=====\n");
//
//    printf("VT = (matlab base-1)\n");
//    printMatrix(n, n, VT, lda, "VT");
//    printf("=====\n");
//
//// step 5: measure error of singular value
//    double ds_sup = 0;
//    for(int j = 0; j < n; j++){
//        double err = fabs( S[j] - S_exact[j] );
//        ds_sup = (ds_sup > err)? ds_sup : err;
//    }
//    printf("|S - S_exact| = %E \n", ds_sup);
//
//// step 6: |A - U*S*VT|
//    // W = S*VT
//    cublas_status = hipblasDdgmm(
//        cublasH,
//        HIPBLAS_SIDE_LEFT,
//        n,
//        n,
//        d_VT,
//        lda,
//        d_S,
//         1,
//        d_W,
//        lda);
//    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
//
//      // A := -U*W + A
//    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*n, hipMemcpyHostToDevice);
//    assert(hipSuccess == cudaStat1);
//    cublas_status = hipblasDgemm(
//        cublasH,
//        HIPBLAS_OP_N, // U
//        HIPBLAS_OP_N, // W
//        m, // number of rows of A
//        n, // number of columns of A
//        n, // number of columns of U
//        &h_minus_one, /* host pointer */
//        d_U, // U
//        lda,
//        d_W, // W
//        lda,
//        &h_one, /* hostpointer */
//        d_A,
//        lda);
//    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
//
//    double dR_fro = 0.0;
//    cublas_status = hipblasDnrm2(
//        cublasH, lda*n, d_A, 1, &dR_fro);
//    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
//
//    printf("|A - U*S*VT| = %E \n", dR_fro);
//
//    // pinv(A) = V*S^-1*UT
//
//
//// free resources
//    if (d_A    ) hipFree(d_A);
//    if (d_S    ) hipFree(d_S);
//    if (d_U    ) hipFree(d_U);
//    if (d_VT   ) hipFree(d_VT);
//    if (devInfo) hipFree(devInfo);
//    if (d_work ) hipFree(d_work);
//    if (d_rwork) hipFree(d_rwork);
//    if (d_W    ) hipFree(d_W);
//
//    if (cublasH ) hipblasDestroy(cublasH);
//    if (cusolverH) hipsolverDnDestroy(cusolverH);
//
//    hipDeviceReset();
//
//    return 0;
//}
