#include "hip/hip_runtime.h"
#include "darkroom/LighthouseEstimator.cuh"

LighthouseEstimatorCUDA::LighthouseEstimatorCUDA() {
    if (!ros::isInitialized()) {
        int argc = 0;
        char **argv = NULL;
        ros::init(argc, argv, "LighthouseEstimatorCUDA",
                  ros::init_options::NoSigintHandler | ros::init_options::AnonymousName);
    }
    nh = ros::NodeHandlePtr(new ros::NodeHandle);
    sensor_location_pub = nh->advertise<roboy_communication_middleware::DarkRoomSensor>(
            "/roboy/middleware/DarkRoom/sensor_location", 1);
    lighthouse_pose_correction = nh->advertise<roboy_communication_middleware::LighthousePoseCorrection>(
            "/roboy/middleware/DarkRoom/LighthousePoseCorrection", 1);
    spinner = boost::shared_ptr<ros::AsyncSpinner>(new ros::AsyncSpinner(1));
    spinner->start();

    tracking = false;
    calibrating = false;
    poseestimating = false;
    distances = false;
    rays = false;
    particle_filtering = false;
    use_lighthouse_calibration_data_phase[LIGHTHOUSE_A] = false;
    use_lighthouse_calibration_data_phase[LIGHTHOUSE_B] = false;
    use_lighthouse_calibration_data_tilt[LIGHTHOUSE_A] = false;
    use_lighthouse_calibration_data_tilt[LIGHTHOUSE_B] = false;
    use_lighthouse_calibration_data_gibphase[LIGHTHOUSE_A] = false;
    use_lighthouse_calibration_data_gibphase[LIGHTHOUSE_B] = false;
    use_lighthouse_calibration_data_gibmag[LIGHTHOUSE_A] = false;
    use_lighthouse_calibration_data_gibmag[LIGHTHOUSE_B] = false;

//    object_pose = VectorXd(6);
//    object_pose << 0, 0, 0, 0, 0, 0.001;
}

void LighthouseEstimatorCUDA::getVisibleCalibratedSensors(vector<int> &visible_sensors) {
    for (auto &sensor : sensors) {
        if (sensor.second.isActive(LIGHTHOUSE_A) &&
            sensor.second.isActive(LIGHTHOUSE_B) &&
            sensor.second.isCalibrated())
            visible_sensors.push_back(sensor.first);
    }
}

void LighthouseEstimatorCUDA::getVisibleCalibratedSensors(bool lighthouse, vector<int> &visible_sensors) {
    for (auto &sensor : sensors) {
        if (sensor.second.isActive(lighthouse) && sensor.second.isCalibrated())
            visible_sensors.push_back(sensor.first);
    }
}

bool LighthouseEstimatorCUDA::estimateSensorPositionsUsingRelativeDistances(bool lighthouse, vector<int> &specificIds) {
    ROS_INFO_STREAM("estimating distance of sensors to lighthouse " << lighthouse + 1);
    vector<Vector3d> relPos;
    vector<double> elevations, azimuths;
    vector<double> distanceToLighthouse;
    vector<int> ids;
    if (specificIds.empty()) {
        // let's see who is active
        cout << "using sensors:" << endl;
        for (auto &sensor : sensors) {
            // skip inactive/uncalibrated sensors
            if (sensor.second.isActive(lighthouse) && sensor.second.isCalibrated()) {
                ids.push_back(sensor.first);
                sensor.second.get(lighthouse, elevations, azimuths);
                // apply factory calibration correction if desired
                applyCalibrationData(lighthouse, elevations.back(), azimuths.back());
                sensor.second.getRelativeLocation(relPos);
                distanceToLighthouse.push_back(sensor.second.getDistance(lighthouse));
                cout << sensor.first << "\t";
            }
        }
        cout << endl;
    } else {
        uint sensor_counter = 0;
        for (uint i = 0; i < specificIds.size(); i++) {
            // skip inactive sensors
            if (sensors[specificIds.at(i)].isActive(lighthouse)) {
                sensor_counter++;
            } else {
                ROS_WARN_THROTTLE(1, "sensor%d inactive", specificIds.at(i));
            }
        }
        if (sensor_counter < specificIds.size()) {
            ROS_WARN("time out waiting for specific sensors");
            return false;
        }
        // get the values now that all requested sensors are active
        for (uint i = 0; i < specificIds.size(); i++) {
            ids.push_back(specificIds.at(i));
            sensors[specificIds.at(i)].get(lighthouse, elevations, azimuths);
            // apply factory calibration correction if desired
            // apply factory calibration correction if desired
            applyCalibrationData(lighthouse, elevations.back(), azimuths.back());
            sensors[specificIds.at(i)].getRelativeLocation(relPos);
            distanceToLighthouse.push_back(sensors[specificIds.at(i)].getDistance(lighthouse));
        }
    }

    if (ids.size() < 3)
        return false;
    // cost function
    auto f = [](double &R0, double &R1, double &cosine, double &distance) {
        return (pow(R0, 2.0) + pow(R1, 2.0) - 2.0 * R0 * R1 * cosine - pow(distance, 2.0));
    };
    // partial derivative
    auto df = [](double &R0, double &R1, double &cosine) { return (2.0 * R0 - 2 * R1 * cosine); };

    MatrixXd cosineBetween(ids.size(), ids.size()), distanceBetween(ids.size(), ids.size());
    for (uint i = 0; i < ids.size() - 1; i++) {
        for (uint j = i + 1; j < ids.size(); j++) {
            // calculate the cosine between the two sensors
            cosineBetween(i, j) =
                    sin(elevations[i]) * cos(azimuths[i]) * sin(elevations[j]) * cos(azimuths[j]) +
                    sin(elevations[i]) * sin(azimuths[i]) * sin(elevations[j]) * sin(azimuths[j]) +
                    cos(elevations[i]) * cos(elevations[j]);

            ROS_DEBUG("cosine between %d and %d: %f", i, j, cosineBetween(i, j));
            // calculate the distance between the sensors
            distanceBetween(i, j) = (relPos[i] - relPos[j]).norm();
            ROS_DEBUG("distance between sensor %d and %d: %f", ids[i], ids[j], distanceBetween(i, j));
        }
    }

    int iterations = 0;

    uint n = ids.size();
    MatrixXd J(n * (n - 1) / 2, n);
    J = J.setZero();
    VectorXd v(n * (n - 1) / 2), d_old(ids.size());

    double error, error_prev = 10000000;
    while (iterations < MAX_ITERATIONS) {

        // construct jacobian and function vector
        int row = 0;
        for (uint i = 0; i < ids.size() - 1; i++) {
            for (uint j = i + 1; j < ids.size(); j++) {
                J(row, i) = df(distanceToLighthouse[i], distanceToLighthouse[j], cosineBetween(i, j));
                J(row, j) = df(distanceToLighthouse[j], distanceToLighthouse[i], cosineBetween(i, j));
                v(row) = f(distanceToLighthouse[i], distanceToLighthouse[j], cosineBetween(i, j),
                           distanceBetween(i, j));
                row++;
            }
        }
        for (uint i = 0; i < ids.size(); i++) {
            d_old(i) = distanceToLighthouse[i];
        }
//        if (iterations % 100 == 0) {
//            ROS_INFO_STREAM("J\n" << J);
//            ROS_INFO_STREAM("v\n" << v);
//            ROS_INFO_STREAM("d_old\n" << d_old);
//        }

        error = v.norm() / (double) ids.size();
        if (error < ERROR_THRESHOLD || (error_prev - error) < 0.00000001) {
            break;
        }
        error_prev = error;
        ROS_INFO_THROTTLE(5, "iteration %d error %lf", iterations, error);
        // construct distance new vector, sharing data with the stl container
        Map<VectorXd> d_new(distanceToLighthouse.data(), distanceToLighthouse.size());
//        d_new = d_old - (J.transpose() * J).inverse() * J.transpose() * v;
        iterations++;
    }

    uint i = 0;
    for (auto id:ids) {
        ROS_DEBUG_STREAM("sensor:" << id << " distance to lighthouse " << lighthouse << ": " << d_old(i));

        Vector2d angles(elevations[i], azimuths[i]);
        Eigen::Vector3d u0;
        rayFromLighthouseAngles(angles, u0, lighthouse);

        Vector3d relLocation(d_old(i) * u0(0), d_old(i) * u0(1), d_old(i) * u0(2));
        sensors[id].set(lighthouse, relLocation);

        i++;

        char str[100];
        sprintf(str, "sensor_%d_estimated", id);

        publishSphere(relLocation, (lighthouse ? "lighthouse2" : "lighthouse1"), str,
                      getMessageID(DISTANCE, id, lighthouse), COLOR(0, 1, lighthouse ? 0 : 1, 0.3), 0.01f, 0);

        sprintf(str, "ray_%d", id);
        Vector3d pos(0, 0, 0);
        publishRay(pos, relLocation, (lighthouse ? "lighthouse2" : "lighthouse1"), str,
                   getMessageID(RAY, id, lighthouse), COLOR(0, 1, lighthouse ? 0 : 1, 0.3), 0);

        sprintf(str, "%d", id);
        publishText(relLocation, str, (lighthouse ? "lighthouse2" : "lighthouse1"), "sensor_id", rand(),
                    COLOR(1, 0, 0, 0.5), 0, 0.04f);
    }

    for (auto id:ids) {
        for (auto id2:ids) {
            if (id2 != id) {
                Vector3d pos1, pos2, dir;
                sensors[id].get(lighthouse, pos1);
                sensors[id2].get(lighthouse, pos2);
                dir = pos2 - pos1;
                publishRay(pos1, dir, (lighthouse ? "lighthouse2" : "lighthouse1"), "distance",
                           rand(), COLOR(0, 1, lighthouse ? 0 : 1, 0.5), 0);

                if (distances) {
                    char str[100];
                    sprintf(str, "%.3f", dir.norm());
                    Vector3d pos = pos1 + dir / 2.0;
                    publishText(pos, str, (lighthouse ? "lighthouse2" : "lighthouse1"), "distance", rand(),
                                COLOR(1, 0, 0, 0.5), 0, 0.02f);
                }
            }
        }
    }

    if (iterations < MAX_ITERATIONS)
        if (error < ERROR_THRESHOLD) {
            ROS_WARN_STREAM(
                    "mean squared error " << error << " below threshold " << ERROR_THRESHOLD << " in " << iterations
                                          << " iterations for lighthouse " << lighthouse + 1);
        } else if ((error - error_prev) < 0.00000001) {
            ROS_WARN_STREAM(
                    "mean squared error " << error << " previous " << error_prev << " doesn't get lower after "
                                          << iterations << " iterations for lighthouse " << lighthouse + 1
            );
        } else
            ROS_WARN_STREAM(
                    "maximal number of iterations reached, mean squared error " << error << " in " << iterations
                                                                                << " iterations for lighthouse "
                                                                                << lighthouse + 1
            );
    return true;
}

void LighthouseEstimatorCUDA::triangulateSensors() {
//    high_resolution_clock::time_point timestamp_new[4];
//    map<int, high_resolution_clock::time_point[4]> timestamps_old;
//
//    ros::Rate rate(30);
//    bool lighthouse_active[2];
//    while (tracking) {
//        roboy_communication_middleware::DarkRoomSensor msg;
//
//        Matrix4d RT_0, RT_1;
//        if (!getTransform(LIGHTHOUSE_A, "world", RT_0)) {
//            rate.sleep(); // no need to query for frame faster than it is published
//            continue;
//        }
//        if (!getTransform(LIGHTHOUSE_B, "world", RT_1)) {
//            rate.sleep(); // no need to query for frame faster than it is published
//            continue;
//        }
//
//        int active_sensors_counter = 0;
//
//        for (auto &sensor : sensors) {
//            lighthouse_active[LIGHTHOUSE_A] = sensor.second.isActive(LIGHTHOUSE_A);
//            lighthouse_active[LIGHTHOUSE_B] = sensor.second.isActive(LIGHTHOUSE_B);
//            if (sensor.second.hasNewData(timestamps_old[sensor.first])) {
//                Vector2d lighthouse0_angles;
//                Vector2d lighthouse1_angles;
//                sensor.second.get(LIGHTHOUSE_A, lighthouse0_angles, &timestamp_new[LIGHTHOUSE_A * 2]);
//                sensor.second.get(LIGHTHOUSE_B, lighthouse1_angles, &timestamp_new[LIGHTHOUSE_B * 2]);
//
//                memcpy(timestamps_old[sensor.first], timestamp_new, sizeof(timestamp_new));
//
//                Vector3d ray0, ray1;
//
//                if (lighthouse_active[LIGHTHOUSE_A] && lighthouse_active[LIGHTHOUSE_B]) {
//                    active_sensors_counter++;
//
//                    Vector3d triangulated_position;
//
//                    applyCalibrationData(lighthouse0_angles, lighthouse1_angles);
//
//                    triangulateFromLighthouseAngles(lighthouse0_angles, lighthouse1_angles, RT_0, RT_1,
//                                                    triangulated_position, ray0,
//                                                    ray1);
//
//                    sensor.second.set(triangulated_position);
//
//                    if (!triangulated_position.hasNaN()) {
//                        char str[100], str2[2];
//                        sprintf(str, "sensor_%d", sensor.first);
//                        publishSphere(triangulated_position, "world", str,
//                                      getMessageID(TRIANGULATED, sensor.first), COLOR(0, 1, 0, 0.8), 0.01f, 1);
//                        sprintf(str2, "%d", sensor.first);
//                        publishText(triangulated_position, str2, "world", str, getMessageID(SENSOR_NAME, sensor.first),
//                                    COLOR(1, 1, 1, 0.7), 0.1, 0.04f);
//                        msg.ids.push_back(sensor.first);
//                        geometry_msgs::Vector3 v;
//                        v.x = triangulated_position[0];
//                        v.y = triangulated_position[1];
//                        v.z = triangulated_position[2];
//                        msg.position.push_back(v);
//                    }
//
//                    if (rays) {
//                        Vector3d pos(0, 0, 0);
//                        ray0 *= 5;
//                        publishRay(pos, ray0, "lighthouse1", "rays_lighthouse_1", getMessageID(RAY, sensor.first, 0),
//                                   COLOR(0, 1, 0, 1.0), 1);
//                        ray1 *= 5;
//                        publishRay(pos, ray1, "lighthouse2", "rays_lighthouse_2", getMessageID(RAY, sensor.first, 1),
//                                   COLOR(0, 1, 0, 1.0), 1);
//
//                    }
//                }
//
//                if (distances) {
//                    int id = 0;
//                    for (auto &sensor_other : sensors) {
//                        if (sensor.first != sensor_other.first &&
//                            (sensor_other.second.isActive(0) && sensor_other.second.isActive(1))) {
//                            Vector3d pos1, pos2, dir;
//                            sensor_other.second.getPosition3D(pos2);
//                            sensor.second.getPosition3D(pos1);
//                            dir = pos2 - pos1;
//                            publishRay(pos1, dir, "world", "distance",
//                                       getMessageID(DISTANCES, id++), COLOR(0, 1, 1, 1.0), 1);
//
//                            char str[100];
//                            sprintf(str, "%.3f", dir.norm());
//                            Vector3d pos = pos1 + dir / 2.0;
//                            publishText(pos, str, "world", "distance", getMessageID(DISTANCES, id++),
//                                        COLOR(1, 0, 0, 0.5), 1, 0.02f);
//                        }
//                    }
//                }
//            }
//        }
//        active_sensors = active_sensors_counter;
//        if (msg.ids.size() > 0)
//            sensor_location_pub.publish(msg);
//    }
}

void LighthouseEstimatorCUDA::publishRays() {
//    high_resolution_clock::time_point timestamp_new[4];
//    map<int, high_resolution_clock::time_point[4]> timestamps_old;
//
//    ros::Rate rate(30);
//    bool lighthouse_active[2];
//
//    while (rays) {
//        for (auto &sensor : sensors) {
//            lighthouse_active[LIGHTHOUSE_A] = sensor.second.isActive(LIGHTHOUSE_A);
//            lighthouse_active[LIGHTHOUSE_B] = sensor.second.isActive(LIGHTHOUSE_B);
//            if (sensor.second.hasNewData(timestamps_old[sensor.first])) {
//                Vector2d lighthouse0_angles;
//                Vector2d lighthouse1_angles;
//                sensor.second.get(LIGHTHOUSE_A, lighthouse0_angles, &timestamp_new[LIGHTHOUSE_A * 2]);
//                sensor.second.get(LIGHTHOUSE_B, lighthouse1_angles, &timestamp_new[LIGHTHOUSE_B * 2]);
//
//                memcpy(timestamps_old[sensor.first], timestamp_new, sizeof(timestamp_new));
//
//                if (lighthouse_active[LIGHTHOUSE_A]) {
//                    Vector3d ray;
//                    rayFromLighthouseAngles(lighthouse0_angles, ray, LIGHTHOUSE_A);
//                    Vector3d pos(0, 0, 0);
//                    ray *= 5;
//                    publishRay(pos, ray, "lighthouse1", "rays_lighthouse_1", getMessageID(RAY, sensor.first, 0),
//                               COLOR(1, 0, 0, 0.5), 1);
//                }
//                if (lighthouse_active[LIGHTHOUSE_B]) {
//                    Vector3d ray;
//                    rayFromLighthouseAngles(lighthouse1_angles, ray, LIGHTHOUSE_B);
//                    Vector3d pos(0, 0, 0);
//                    ray *= 5;
//                    publishRay(pos, ray, "lighthouse2", "rays_lighthouse_2", getMessageID(RAY, sensor.first, 0),
//                               COLOR(1, 0, 0, 0.5), 1);
//                }
//            }
//        }
//        rate.sleep();
//    }
}

int LighthouseEstimatorCUDA::getMessageID(int type, int sensor, bool lighthouse) {
//    TRIANGULATED = 0,      // for each sensor
//    DISTANCE = 1,           // for each sensor and lighthouse
//    RAY = 2,   // for each sensor and lighthouse
//    SENSOR_NAME = 3,   // for each sensor
//    DISTANCES = 4
    int n_sensors = sensors.size(), per_lighthouse = n_sensors * NUMBER_OF_LIGHTHOUSES * lighthouse;


    switch (type) {
        case TRIANGULATED:
            return sensor;
        case DISTANCE:
            return n_sensors + per_lighthouse + sensor;
        case RAY:
            return n_sensors + n_sensors * NUMBER_OF_LIGHTHOUSES + n_sensors + per_lighthouse + sensor;
        case SENSOR_NAME:
            return n_sensors + 2 * n_sensors * NUMBER_OF_LIGHTHOUSES + sensor;
        case DISTANCES:
            return 6000 + sensor;
        default:
            return rand();
    }
}

void LighthouseEstimatorCUDA::applyCalibrationData(Vector2d &lighthouse0_angles, Vector2d &lighthouse1_angles) {
    applyCalibrationData(LIGHTHOUSE_A, lighthouse0_angles);
    applyCalibrationData(LIGHTHOUSE_B, lighthouse1_angles);
}

void LighthouseEstimatorCUDA::applyCalibrationData(bool lighthouse, Vector2d &lighthouse_angles) {
    if (use_lighthouse_calibration_data_phase[lighthouse]) {
        lighthouse_angles[HORIZONTAL] += ootx[lighthouse].fcal_0_phase;
        lighthouse_angles[VERTICAL] += ootx[lighthouse].fcal_1_phase;
    }
    if (use_lighthouse_calibration_data_tilt[lighthouse]) {
        lighthouse_angles[HORIZONTAL] += ootx[lighthouse].fcal_0_tilt;
        lighthouse_angles[VERTICAL] += ootx[lighthouse].fcal_1_tilt;
    }
    if (use_lighthouse_calibration_data_gibphase[lighthouse]) {
        // TODO
    }
    if (use_lighthouse_calibration_data_gibmag[lighthouse]) {
        // TODO
    }
}

void LighthouseEstimatorCUDA::applyCalibrationData(bool lighthouse, double &elevation, double &azimuth) {
    if (use_lighthouse_calibration_data_phase[lighthouse]) {
        azimuth += ootx[lighthouse].fcal_0_phase;
        elevation += ootx[lighthouse].fcal_1_phase;
    }
    if (use_lighthouse_calibration_data_tilt[lighthouse]) {
        azimuth += ootx[lighthouse].fcal_0_tilt;
        elevation += ootx[lighthouse].fcal_1_tilt;
    }
    if (use_lighthouse_calibration_data_gibphase[lighthouse]) {
        // TODO
    }
    if (use_lighthouse_calibration_data_gibmag[lighthouse]) {
        // TODO
    }
}
